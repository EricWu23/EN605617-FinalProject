#include <iostream>

#include "mse.h"
#include "train.h"
#include "../utils/utils.h"

#define ENABLE_LOG 1
void debug(float *arrayname,int n_sample,int sz_out){

 if ((n_sample+9)<sz_out){
    for(int i=0; i<10;i++){
        std::cout<< arrayname[n_sample+i] <<' ';
        if(i == 9){
          std::cout<<'\n';
        }
    }
 }        
}
inline void CUDAErrorCheck(hipError_t err,const char * name){
 
    if(err!= hipSuccess)
    {
      std::cerr << "ERROR: " <<  name << " (" << err << ")" << std::endl;
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      exit(-1);
    }
}
void train_gpu(Sequential_GPU & seq, float *inp, float *targ, int bs, int n_in,int n_out, int batch_idx,int epoch_idx,int log_interval,int tbs){

    float* inp_shift=inp+batch_idx*bs*n_in;
    float* targ_shft=targ+batch_idx*bs*n_out;

    int sz_out = bs*n_out;
    MSE_GPU mse(sz_out);
    
    int sz_inp = bs*n_in;
    float *cp_inp, *out;
  
    cp_inp = inp_shift;
    seq.forward(cp_inp, out);// after runing lin1.inp, lin1.out,relu1.inp,relu1.out,lin2.inp, and lin2.out will contain the results from forward propogation
    mse.forward(seq.layers.back()->out, targ_shft);// dummy, store the argument passed in as mse.inp (y_hat), mse.out (targ_shft) 

    mse.backward();//update the mse.inp to be dJ/dy_hat. mse.out stores the targ
    seq.update();
    /* clean up temporary memory at the end of each batch*/
    seq.free();

    if(batch_idx%log_interval==0 && ENABLE_LOG){
      seq.forward(cp_inp, out);
      mse._forward(seq.layers.back()->out, targ_shft);// compute the actual loss
      seq.free();
      std::cout << "Training Epoch:"<< epoch_idx << "| [finished size/traing size] : ["<< (batch_idx+1)*bs<<"/"<<tbs<< "] ("<<
      (int)((batch_idx+1)*bs*100.0/tbs)<<"%) | Training Loss:"<< mse.loss[0] << std::endl;
    }
    mse.free();
}