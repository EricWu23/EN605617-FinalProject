#include "hip/hip_runtime.h"
#include "relu.h"


__global__
void relu_forward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (ind < sz_out){
        out[ind] = fmaxf(0, inp[ind]);
    }
}


__global__
void relu_backward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (ind < sz_out){
        inp[ind] = (0 < inp[ind]) * out[ind];
    }
}


ReLU_GPU::ReLU_GPU(int _bs,int _n_in){
    model_type= relu;
    bs=_bs;
    n_in=_n_in;
    n_out=_n_in;
    sz_out = bs*n_out;
    
    n_blocks = (sz_out + block_size - 1) / block_size;
}


void ReLU_GPU::forward(float *_inp, float *_out){
    inp = _inp;
    out = _out;

    relu_forward_gpu<<<n_blocks, block_size>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}


void ReLU_GPU::backward(){    
    relu_backward_gpu<<<n_blocks, block_size>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}

void ReLU_GPU::update_batchsize(int new_bs){
  if(new_bs!=bs){
    bs=new_bs;
    sz_out = bs*n_out;
    n_blocks = (sz_out + block_size - 1) / block_size;
  }
}